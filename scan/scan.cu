#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define NUM_BANKS 32
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)

extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


__global__ void
prescan(int N, int *arr, int *sums)
{
    int block_index = blockIdx.x;
    int tid = threadIdx.x;
    int base_index = blockIdx.x * N;
    extern __shared__ int temp[];

    int i = tid * 2;
    int a = tid;
    int b = tid + N / 2;
    int bankOffsetA = CONFLICT_FREE_OFFSET(a);
    int bankOffsetB = CONFLICT_FREE_OFFSET(b);

    temp[a + bankOffsetA] = arr[base_index + a];
    temp[b + bankOffsetB] = arr[base_index + b];
    for (int d = 1; d < N; d <<= 1) {
        __syncthreads();
        int offset = d << 1;
        if ((i & (offset - 1)) == 0) {
            int ai = i + d - 1;
            int bi = i + offset - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            temp[bi] += temp[ai];
        }
    }
    if (tid == 0) {
        int padded_N = N - 1 + CONFLICT_FREE_OFFSET(N - 1);
        sums[block_index] = temp[padded_N];
        temp[padded_N] = 0;
    }
    for (int d = N >> 1; d > 0; d >>= 1) {
        __syncthreads();
        int offset = d << 1;
        if ((i & (offset - 1)) == 0) {
            int ai = i + d - 1;
            int bi = i + offset - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    arr[base_index + a] = temp[a + bankOffsetA];
    arr[base_index + b] = temp[b + bankOffsetB];
    
}

__global__ void
add(int *a, int *b)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int added = b[blockIdx.x];
    a[index] += added;
}

void exclusive_scan(int* device_start, int length, int* device_result)
{
    /* Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the input and output in device memory,
     * but this is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the input and the output arrays are sized to accommodate the next
     * power of 2 larger than the input.
     */
    int rounded_length = nextPow2(length);
    int threadPerBlock = 128;
    int elementPerBlock = threadPerBlock * 2;
    int blocks = (rounded_length + elementPerBlock - 1) / elementPerBlock;

    int *sums;
    hipMalloc((void**)&sums, blocks * sizeof(int));
    prescan<<<blocks, threadPerBlock, elementPerBlock * sizeof(int)>>>(elementPerBlock, device_result, sums);

    if (blocks > 1) {
        int *sums_prefix = sums;
        exclusive_scan(sums, blocks, sums_prefix);
        add<<<blocks - 1, elementPerBlock>>>(device_result + elementPerBlock, sums_prefix + 1);
    }

    hipFree(sums);
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input; 
    // We round the array sizes up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness. 
    // You may have an easier time in your implementation if you assume the 
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    // For convenience, both the input and output vectors on the device are
    // initialized to the input values. This means that you are free to simply
    // implement an in-place scan on the result vector if you wish.
    // If you do this, you will need to keep that fact in mind when calling
    // exclusive_scan from find_repeats.
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, end - inarray, device_result);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void
tag_repeats_kernel(int *input, int N, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1 && input[index] == input[index + 1])
        output[index] = 1;
    else
        output[index] = 0;
}

__global__ void
filter_kernel(int *input, int N, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1 && input[index] + 1 == input[index + 1])
        output[input[index]] = index;
}


int find_repeats(int *device_input, int length, int *device_output) {
    /* Finds all pairs of adjacent repeated elements in the list, storing the
     * indices of the first element of each pair (in order) into device_result.
     * Returns the number of pairs found.
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if 
     * it requires that. However, you must ensure that the results of
     * find_repeats are correct given the original length.
     */    
    int rounded_length = nextPow2(length);
    int threadPerBlock = 512;
    int blocks = (rounded_length + threadPerBlock - 1) / threadPerBlock;
    int *device_isRepeat;
    hipMalloc((void**)&device_isRepeat, rounded_length * sizeof(int));

    double startTime, endTime;

    startTime = CycleTimer::currentSeconds();
    tag_repeats_kernel<<<blocks, threadPerBlock>>>(device_input, length, device_isRepeat);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("tag time = %.3f\n", 1000.f * (endTime - startTime));

    startTime = CycleTimer::currentSeconds();
    exclusive_scan(device_isRepeat, rounded_length, device_isRepeat);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("scan time = %.3f\n", 1000.f * (endTime - startTime));

    startTime = CycleTimer::currentSeconds();
    filter_kernel<<<blocks, threadPerBlock>>>(device_isRepeat, length, device_output);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("filter time = %.3f\n", 1000.f * (endTime - startTime));

    startTime = CycleTimer::currentSeconds();
    int output_length;
    hipMemcpy(&output_length, &device_isRepeat[length - 1], sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("cpy time = %.3f\n", 1000.f * (endTime - startTime));
    hipFree(device_isRepeat);

    return output_length;
}

/* Timing wrapper around find_repeats. You should not modify this function.
 */
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
